#include "hip/hip_runtime.h"
// Copyright (c) 2017 Doyub Kim
//
// I am making my contributions/submissions to this project solely in my
// personal capacity and am not conveying any rights to any intellectual
// property of any third parties.

#include <jet/cuda_point_hash_grid_searcher3.h>

#include <thrust/for_each.h>

using namespace jet;
using namespace experimental;

namespace {

struct InitializeTables {
    template <typename Tuple>
    inline JET_CUDA_DEVICE void operator()(Tuple t) {
        thrust::get<0>(t) = kMaxSize;
        thrust::get<1>(t) = kMaxSize;
    }
};

struct InitializeIndexPointAndKeys {
    CudaPointHashGridSearcher3::HashUtils hashUtils;

    inline JET_CUDA_HOST_DEVICE InitializeIndexPointAndKeys(float gridSpacing,
                                                            int3 resolution)
        : hashUtils(gridSpacing, resolution) {}

    template <typename Tuple>
    inline JET_CUDA_DEVICE void operator()(Tuple t) {
        // 0: i [in]
        // 1: sortedIndices[out]
        // 2: points[in]
        // 3: points[out]
        // 4: keys[out]
        size_t i = thrust::get<0>(t);
        thrust::get<1>(t) = i;
        float4 p = thrust::get<2>(t);
        thrust::get<3>(t) = p;
        size_t key = hashUtils.getHashKeyFromPosition(p);
        thrust::get<4>(t) = key;
    }
};

struct BuildTables {
    size_t* keys;
    size_t* startIndexTable;
    size_t* endIndexTable;

    inline JET_CUDA_HOST_DEVICE BuildTables(size_t* k, size_t* sit, size_t* eit)
        : keys(k), startIndexTable(sit), endIndexTable(eit) {}

    template <typename Index>
    inline JET_CUDA_DEVICE void operator()(Index i) {
        size_t k = keys[i];
        size_t kLeft = keys[i - 1];
        if (k > kLeft) {
            startIndexTable[k] = i;
            endIndexTable[kLeft] = i;
        }
    }
};

}  // namespace

CudaPointHashGridSearcher3::CudaPointHashGridSearcher3(const Size3& resolution,
                                                       float gridSpacing)
    : CudaPointHashGridSearcher3(resolution.x, resolution.y, resolution.z,
                                 gridSpacing) {}

CudaPointHashGridSearcher3::CudaPointHashGridSearcher3(size_t resolutionX,
                                                       size_t resolutionY,
                                                       size_t resolutionZ,
                                                       float gridSpacing)
    : _gridSpacing(gridSpacing) {
    _resolution.x = std::max(static_cast<int>(resolutionX), 1);
    _resolution.y = std::max(static_cast<int>(resolutionY), 1);
    _resolution.z = std::max(static_cast<int>(resolutionZ), 1);

    _startIndexTable.resize(_resolution.x * _resolution.y * _resolution.z,
                            kMaxSize);
    _endIndexTable.resize(_resolution.x * _resolution.y * _resolution.z,
                          kMaxSize);
}

CudaPointHashGridSearcher3::CudaPointHashGridSearcher3(
    const CudaPointHashGridSearcher3& other) {
    set(other);
}

void CudaPointHashGridSearcher3::build(const CudaArrayView1<float4>& points) {
    _points.clear();
    _keys.clear();
    _startIndexTable.clear();
    _endIndexTable.clear();
    _sortedIndices.clear();

    // Allocate memory chuncks
    size_t numberOfPoints = points.size();
    _startIndexTable.resize(_resolution.x * _resolution.y * _resolution.z);
    _endIndexTable.resize(_resolution.x * _resolution.y * _resolution.z);
    _keys.resize(numberOfPoints);
    _sortedIndices.resize(numberOfPoints);
    _points.resize(numberOfPoints);

    if (numberOfPoints == 0) {
        return;
    }

    // Initialize tables
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(
                         _startIndexTable.begin(), _endIndexTable.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(
                         _startIndexTable.end(), _endIndexTable.end())),
                     InitializeTables());

    // Initialize indices array and generate hash key for each point
    auto countingBegin = thrust::counting_iterator<size_t>(0);
    auto countingEnd = countingBegin + numberOfPoints;
    thrust::for_each(thrust::make_zip_iterator(thrust::make_tuple(
                         countingBegin, _sortedIndices.begin(), points.begin(),
                         _points.begin(), _keys.begin())),
                     thrust::make_zip_iterator(thrust::make_tuple(
                         countingEnd, _sortedIndices.end(), points.end(),
                         _points.end(), _keys.end())),
                     InitializeIndexPointAndKeys(_gridSpacing, _resolution));

    // Sort indices/points/key based on hash key
    thrust::sort_by_key(_keys.begin(), _keys.end(),
                        thrust::make_zip_iterator(thrust::make_tuple(
                            _sortedIndices.begin(), _points.begin())));

    // Now _points and _keys are sorted by points' hash key values.
    // Let's fill in start/end index table with _keys.

    // Assume that _keys array looks like:
    // [5|8|8|10|10|10]
    // Then _startIndexTable and _endIndexTable should be like:
    // [.....|0|...|1|..|3|..]
    // [.....|1|...|3|..|6|..]
    //       ^5    ^8   ^10
    // So that _endIndexTable[i] - _startIndexTable[i] is the number points
    // in i-th table bucket.

    _startIndexTable[_keys[0]] = 0;
    _endIndexTable[_keys[numberOfPoints - 1]] = numberOfPoints;

    thrust::for_each(countingBegin + 1, countingEnd,
                     BuildTables(_keys.data(), _startIndexTable.data(),
                                 _endIndexTable.data()));
}

CudaArrayView1<size_t> CudaPointHashGridSearcher3::keys() const {
    return _keys.view();
}

CudaArrayView1<size_t> CudaPointHashGridSearcher3::startIndexTable() const {
    return _startIndexTable.view();
}

CudaArrayView1<size_t> CudaPointHashGridSearcher3::endIndexTable() const {
    return _endIndexTable.view();
}

CudaArrayView1<size_t> CudaPointHashGridSearcher3::sortedIndices() const {
    return _sortedIndices.view();
}

CudaPointHashGridSearcher3& CudaPointHashGridSearcher3::operator=(
    const CudaPointHashGridSearcher3& other) {
    set(other);
    return (*this);
}

void CudaPointHashGridSearcher3::set(const CudaPointHashGridSearcher3& other) {
    _gridSpacing = other._gridSpacing;
    _resolution = other._resolution;
    _points = other._points;
    _keys = other._keys;
    _startIndexTable = other._startIndexTable;
    _endIndexTable = other._endIndexTable;
    _sortedIndices = other._sortedIndices;
}
